#include "hip/hip_runtime.h"
#include <iostream>
#include "timer.h"
#include "utils.h"

template <typename T>
__global__
void transpose_v5(const T * const d_mat, T * const d_out, const size_t N) 
{
  extern __shared__ __align__(sizeof(T)) unsigned char s_mem[];
  T *s_mat = reinterpret_cast<T *>(s_mem);

  int tix = threadIdx.x, tiy = threadIdx.y;
  int bdx = blockDim.x, bdy = blockDim.y;
  int x = blockIdx.x * bdx + tix;
  int y = blockIdx.y * bdy + tiy;

  if (x >= N || y >= N)
    return;

  s_mat[tiy + (bdy+1) * tix] = d_mat[x + y * N];
  __syncthreads();

  d_out[tix + blockIdx.y * bdy + (tiy + blockIdx.x * bdx) * N] = s_mat[tix + (bdx+1) * tiy];
}

template <typename T>
float parallel_transpose_v5(const T *h_matrix, T *h_output, const size_t N)
{
  T *d_matrix, *d_output;
  const size_t matrixSize = N * N * sizeof(T);
  const dim3 blockSize(16,16,1);
  const dim3 gridSize(round_up(N, (size_t)blockSize.x), round_up(N, (size_t)blockSize.y), 1);

  hipMalloc((void **) &d_matrix, matrixSize);
  hipMalloc((void **) &d_output, matrixSize);
CHECK_ERR
  hipMemcpy(d_matrix, h_matrix, matrixSize, hipMemcpyHostToDevice);

  GpuTimer timer;
  timer.Start();
  transpose_v5<<<gridSize, blockSize, blockSize.x * (blockSize.y + 1) * sizeof(T)>>>(d_matrix, d_output, N);
CHECK_ERR
  timer.Stop();
  hipMemcpy(h_output, d_output, matrixSize, hipMemcpyDeviceToHost);

  hipFree(d_matrix);
  hipFree(d_output);
CHECK_ERR

  return timer.Elapsed();
}

// initialize transpose for certain types
INIT_TRANSPOSE_FUNC(parallel_transpose_v5, ui);
INIT_TRANSPOSE_FUNC(parallel_transpose_v5, ull);
INIT_TRANSPOSE_FUNC(parallel_transpose_v5, float);